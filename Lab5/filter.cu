#include "hip/hip_runtime.h"
// Lab 5, image filters with CUDA.

// Compile with a command-line similar to Lab 4:
// nvcc filter.cu -c -arch=sm_30 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -lcudart -L/usr/local/cuda/lib -lglut -o filter
// or (multicore lab)
// nvcc filter.cu -c -arch=sm_20 -o filter.o
// g++ filter.o milli.c readppm.c -lGL -lm -lcuda -L/usr/local/cuda/lib64 -lcudart -lglut -o filter

// 2017-11-27: Early pre-release, dubbed "beta".
// 2017-12-03: First official version! Brand new lab 5 based on the old lab 6.
// Better variable names, better prepared for some lab tasks. More changes may come
// but I call this version 1.0b2.
// 2017-12-04: Two fixes: Added command-lines (above), fixed a bug in computeImages
// that allocated too much memory. b3
// 2017-12-04: More fixes: Tightened up the kernel with edge clamping.
// Less code, nicer result (no borders). Cleaned up some messed up X and Y. b4

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#ifdef __APPLE__
  #include <GLUT/glut.h>
  #include <OpenGL/gl.h>
#else
  #include <GL/glut.h>
#endif
#include "readppm.h"
#include "milli.h"

// Use these for setting shared memory size.
#define maxKernelSizeX 10
#define maxKernelSizeY 10

//#define separable
__global__ void filter(unsigned char *image, unsigned char *out, const unsigned int imagesizex, const unsigned int imagesizey, const int kernelsizex, const int kernelsizey)
{
  // map from blockIdx to pixel position
  // I.e. original image base coordinate
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // ****** Allocate Shared Memory ****** //
  // We need a MAXMEMSIZE since kernelsize specify a filter of size (2*kernelsize + 1)
  const int MAXMEMSIZEX = 2 * maxKernelSizeX + 1;
  const int MAXMEMSIZEY = 2 * maxKernelSizeY + 1;
  __shared__ unsigned char smem[MAXMEMSIZEX*MAXMEMSIZEY*3]; // 3 for RGB

  // Define our shared memory block
  // (Avoid using branching by using max and min)
  // ***** THE PROBLEM SEEMS TO BE LOCATED HERE ***** //
  int memBlockStartX = max(0, (int)(blockIdx.x*blockDim.x) - kernelsizex);
  int memBlockStartY = max(0, (int)(blockIdx.y*blockDim.y) - kernelsizey);
  int memBlockEndX = min(imagesizex-1, memBlockStartX + (int)blockDim.x + (2*kernelsizex -1)); // Using different constants in the last paranthesis seems to alter the result the most
  int memBlockEndY = min(imagesizey-1, memBlockStartY + (int)blockDim.y + (2*kernelsizey -1)); // These values provide a nice result for separable filters though...

  // Define thread memory by calculating shared memory block size to actual block size ratio
  int memBlockSize = (memBlockEndX - memBlockStartX + 1) * (memBlockEndY - memBlockStartY + 1);
  int blocksize = blockDim.x * blockDim.y;
  int threadMem = (int)(memBlockSize/(blocksize));

  int memSizeX = memBlockEndX - memBlockStartX + 1;

  // Load the ammount of pixel memory allowed for each thread to shared memory
  for(int i = 0; i <= threadMem; i++) {// TODO: Find corresponding image data to memory index, no RGB?
    // (Remember, our Shared Memory is a 1D array)
    // Traverse our shared memory block
    int memIndex = (threadIdx.x + threadIdx.y * memSizeX + i * blocksize);
    int memCurrentX = memIndex % memSizeX;
    int memCurrentY = (int)((memIndex - memCurrentX) / memSizeX);
    // TODO: Add RGB functionality
    memIndex *= 3;

    // Map to image index
    int imgX = memBlockStartX + memCurrentX;
    int imgY = memBlockStartY + memCurrentY;
    int imgIndex = 3 * (imgX + imgY * imagesizex);

    if( memIndex <=  3 * memBlockSize ) {

      smem[memIndex+0] = image[imgIndex];
      smem[memIndex+1] = image[imgIndex+1];
      smem[memIndex+2] = image[imgIndex+2];
    }
  }

  __syncthreads();

  // ****** Actual Filter ****** //
  int dy, dx;
  unsigned int sumx, sumy, sumz;

  int divby = (2*kernelsizex+1) * (2*kernelsizex+1); // Works for box filters only!

  // Shared Memory coordinates
  int sx = x - memBlockStartX;
  int sy = y - memBlockStartY;

	if (x < imagesizex && y < imagesizey) // If inside image
	{
  // Filter kernel (simple box filter)
	sumx=0;sumy=0;sumz=0;
	for(dy=-kernelsizey;dy<=kernelsizey;dy++)
		for(dx=-kernelsizex;dx<=kernelsizex;dx++)
		{
			// Use max and min to avoid branching!
      int xx = min(max(sx+dx, 0), memBlockEndX);
      int yy = min(max(sy+dy, 0), memBlockEndY);

      int sharedIndex = 3* (xx + memSizeX*yy);

      // Instead, collect data from Shared Memory rather than Global Memory
			sumx += smem[sharedIndex];
			sumy += smem[sharedIndex+1];
			sumz += smem[sharedIndex+2];
		}
	out[(y*imagesizex+x)*3+0] = sumx/divby;
	out[(y*imagesizex+x)*3+1] = sumy/divby;
	out[(y*imagesizex+x)*3+2] = sumz/divby;
	}
}

// Global variables for image data
unsigned char *image, *pixels, *dev_bitmap, *dev_input, *dev_temp;
unsigned int imagesizey, imagesizex; // Image size

////////////////////////////////////////////////////////////////////////////////
// MAIN COMPUTATION FUNCTION
////////////////////////////////////////////////////////////////////////////////
void computeImages(int kernelsizex, int kernelsizey)
{
	if (kernelsizex > maxKernelSizeX || kernelsizey > maxKernelSizeY)
	{
		printf("Kernel size out of bounds!\n");
		return;
	}

  // ***** OUR BLOCKSIZE VARIABLE IS PROVIDING SOME WEIRD OUTPUTS IF CHANGED AS WELL ****** //
  // For boxfilters we cannot use a blocksize >= 10
  int blocksize = 10;

	pixels = (unsigned char *) malloc(imagesizex*imagesizey*3);
	hipMalloc( (void**)&dev_input, imagesizex*imagesizey*3);
	hipMemcpy( dev_input, image, imagesizey*imagesizex*3, hipMemcpyHostToDevice );
	hipMalloc( (void**)&dev_bitmap, imagesizex*imagesizey*3);

  hipMalloc( (void**)&dev_temp, imagesizex * imagesizey * 3);
    // If we want to use separable filter kernels, run this code
    #ifdef separable
      dim3 grid1(imagesizex/(blocksize), imagesizey);
      dim3 grid2(imagesizex*3, imagesizey/blocksize);
      dim3 blockGrid1(blocksize,1);
      dim3 blockGrid2(3*1, blocksize);
      filter<<<grid1, blockGrid1>>>(dev_input, dev_temp, imagesizex, imagesizey, kernelsizex, 0);   // Output goes into temp variable, no kernelsizey
      filter<<<grid2, blockGrid2>>>(dev_temp, dev_bitmap, imagesizex, imagesizey, 0, kernelsizey);  // Input is temp variable here, no kernelsizex
    #else
    // "Normal" box-filter kernel
      dim3 grid(3*imagesizex/ blocksize, imagesizey / blocksize);
      dim3 blockGrid(3*blocksize, blocksize);
    //  dim3 grid(imagesizex, imagesizey);
      filter<<<grid, blockGrid>>>(dev_input, dev_bitmap, imagesizex, imagesizey, kernelsizex, kernelsizey); // Awful load balance
    #endif

	hipDeviceSynchronize();
//	Check for errors!
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(err));
	hipMemcpy( pixels, dev_bitmap, imagesizey*imagesizex*3, hipMemcpyDeviceToHost );
	hipFree( dev_bitmap );
	hipFree( dev_input );
  #ifdef separable
    hipFree( dev_temp);
  #endif
}

// Display images
void Draw()
{
// Dump the whole picture onto the screen.
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );

	if (imagesizey >= imagesizex)
	{ // Not wide - probably square. Original left, result right.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
		glRasterPos2i(0, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE,  pixels);
	}
	else
	{ // Wide image! Original on top, result below.
		glRasterPos2f(-1, -1);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, pixels );
		glRasterPos2i(-1, 0);
		glDrawPixels( imagesizex, imagesizey, GL_RGB, GL_UNSIGNED_BYTE, image );
	}
	glFlush();
}

// Main program, inits
int main( int argc, char** argv)
{
  printf("\n*----------- PROGRAM INFO -----------* \n\n");
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );

	if (argc > 1)
		image = readppm(argv[1], (int *)&imagesizex, (int *)&imagesizey);
	else
		image = readppm((char *)"maskros512.ppm", (int *)&imagesizex, (int *)&imagesizey);

	if (imagesizey >= imagesizex)
		glutInitWindowSize( imagesizex*2, imagesizey );
	else
		glutInitWindowSize( imagesizex, imagesizey*2 );
	glutCreateWindow("Lab 5");
	glutDisplayFunc(Draw);

	ResetMilli();
	computeImages(2, 2);
  int time = GetMicroseconds();

  printf("\n*----------- BENCHMARKING -----------*");
	printf("\n\nFiltering took %i microseconds. \n\n", time );

// You can save the result to a file like this:
  writeppm("out.ppm", imagesizey, imagesizex, pixels);

	glutMainLoop();
	return 0;
}
